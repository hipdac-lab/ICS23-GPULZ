#include "hip/hip_runtime.h"
#include "utils.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <cmath>
#include <hipcub/hipcub.hpp>

#define BLOCK_SIZE 2048     // in unit of vector, the size of one data block
#define THREAD_SIZE 128     // in unit of vector, the size of the thread block
#define WINDOW_SIZE 32      // in unit of datatype, maximum 255, the size of the sliding window, so as the maximum match length
#define INPUT_TYPE uint32_t // define input type, since c++ doesn't support runtime data type defination
#define VECTOR_SIZE 16      // in unit of datatype (uint32_t), the size of the vector
// #define DEBUG

// each thread block handles VECTOR_SIZE * THREAD_SIZE = 2048 data points

__device__ bool vectorComparison(INPUT_TYPE *input, uint32_t vectorSize, uint32_t bufferPosition, uint32_t windowPosition)
{
    for (int tmpIdx = 0; tmpIdx < vectorSize; tmpIdx++)
    {
        if (input[bufferPosition + tmpIdx] != input[windowPosition + tmpIdx])
        {
            return false;
        }
    }
    return true;
}

// Define the compress match kernel functions
template <typename T>
__global__ void compressKernelI(T *input, uint32_t numOfBlocks, uint32_t *flagArrSizeGlobal, uint32_t *compressedDataSizeGlobal, uint8_t *tmpFlagArrGlobal, uint8_t *tmpCompressedDataGlobal, int minEncodeLength)
{
    // Block size in uint of datatype
    const uint32_t blockSize = BLOCK_SIZE;

    // Window size in uint of datatype
    const uint32_t threadSize = THREAD_SIZE;

    const uint32_t vectorSize = VECTOR_SIZE;

    // Allocate shared memory for the lookahead buffer information
    __shared__ uint8_t lengthBuffer[blockSize];
    __shared__ uint8_t offsetBuffer[blockSize];
    __shared__ uint8_t byteFlagArr[(blockSize / 8)];
    __shared__ uint32_t prefixBuffer[blockSize + 1];

    // initialize the start position, in unit of vector
    int startPosision = blockIdx.x * blockSize;

    int vectorIdx = 0;

    // find match for every data point
    for (int iteration = 0; iteration < (int)(blockSize / threadSize); iteration++)
    {
        // Initialize the lookahead buffer and the sliding window pointers
        // initialize the vectorIdx, in unit of vector
        vectorIdx = threadIdx.x + iteration * threadSize;
        int bufferStart = vectorIdx;
        int bufferPointer = bufferStart;
        int windowStart = bufferStart - int(WINDOW_SIZE) < 0 ? 0 : bufferStart - WINDOW_SIZE;
        int windowPointer = windowStart;

        uint8_t maxLen = 0;
        uint8_t maxOffset = 0;
        uint8_t len = 0;
        uint8_t offset = 0;

        while (windowPointer < bufferStart && bufferPointer < blockSize)
        {
            if (vectorComparison(input, vectorSize, (startPosision + bufferPointer) * vectorSize, (startPosision + windowPointer) * vectorSize))
            {
                if (offset == 0)
                {
                    offset = bufferPointer - windowPointer;
                }
                len++;
                bufferPointer++;
            }
            else
            {
                if (len > maxLen)
                {
                    maxLen = len;
                    maxOffset = offset;
                }
                len = 0;
                offset = 0;
                bufferPointer = bufferStart;
            }
            windowPointer++;
        }
        if (len > maxLen)
        {
            maxLen = len;
            maxOffset = offset;
        }

        lengthBuffer[vectorIdx] = maxLen;
        offsetBuffer[vectorIdx] = maxOffset;

        // initialize array as 0
        prefixBuffer[vectorIdx] = 0;
    }
    __syncthreads();

    // find encode information
    uint32_t flagCount = 0;

    if (threadIdx.x == 0)
    {
        uint8_t flagPosition = 0x01;
        uint8_t byteFlag = 0;

        int encodeIndex = 0;

        while (encodeIndex < blockSize)
        {
            // if length < minEncodeLength, no match is found
            if (lengthBuffer[encodeIndex] < minEncodeLength)
            {
                prefixBuffer[encodeIndex] = vectorSize * sizeof(T);
                encodeIndex++;
            }
            // if length > minEncodeLength, match is found
            else
            {
                prefixBuffer[encodeIndex] = 2;
                encodeIndex += lengthBuffer[encodeIndex];
                byteFlag |= flagPosition;
            }
            // store the flag if there are 8 bits already
            if (flagPosition == 0x80)
            {
                byteFlagArr[flagCount] = byteFlag;
                flagCount++;
                flagPosition = 0x01;
                byteFlag = 0;
                continue;
            }
            flagPosition <<= 1;
        }
        if (flagPosition != 0x01)
        {
            byteFlagArr[flagCount] = byteFlag;
            flagCount++;
        }
    }
    __syncthreads();

    // prefix summation, up-sweep
    int prefixSumOffset = 1;
    for (uint32_t d = blockSize >> 1; d > 0; d = d >> 1)
    {
        for (int iteration = 0; iteration < (int)(blockSize / threadSize); iteration++)
        {
            vectorIdx = threadIdx.x + iteration * threadSize;
            if (vectorIdx < d)
            {
                int ai = prefixSumOffset * (2 * vectorIdx + 1) - 1;
                int bi = prefixSumOffset * (2 * vectorIdx + 2) - 1;
                prefixBuffer[bi] += prefixBuffer[ai];
            }
            __syncthreads();
        }
        prefixSumOffset *= 2;
    }

    // clear the last element
    if (threadIdx.x == 0)
    {
        // printf("block size: %d flag array size: %d\n", prefixBuffer[blockSize - 1], flagCount);
        compressedDataSizeGlobal[blockIdx.x] = prefixBuffer[blockSize - 1];
        flagArrSizeGlobal[blockIdx.x] = flagCount;
        prefixBuffer[blockSize] = prefixBuffer[blockSize - 1];
        prefixBuffer[blockSize - 1] = 0;
    }
    __syncthreads();

    // prefix summation, down-sweep
    for (int d = 1; d < blockSize; d *= 2)
    {
        prefixSumOffset >>= 1;
        for (int iteration = 0; iteration < (int)(blockSize / threadSize);
             iteration++)
        {
            vectorIdx = threadIdx.x + iteration * threadSize;

            if (vectorIdx < d)
            {
                int ai = prefixSumOffset * (2 * vectorIdx + 1) - 1;
                int bi = prefixSumOffset * (2 * vectorIdx + 2) - 1;

                uint32_t t = prefixBuffer[ai];
                prefixBuffer[ai] = prefixBuffer[bi];
                prefixBuffer[bi] += t;
            }
            __syncthreads();
        }
    }

    // encoding phase one
    int blockOffset = blockSize * blockIdx.x;

    for (int iteration = 0; iteration < (int)(blockSize / threadSize); iteration++)
    {
        vectorIdx = threadIdx.x + iteration * threadSize;
        if (prefixBuffer[vectorIdx + 1] != prefixBuffer[vectorIdx])
        {
            if (lengthBuffer[vectorIdx] < minEncodeLength)
            {
                uint32_t tmpOffset = prefixBuffer[vectorIdx];
                // uint8_t *bytePtr = (uint8_t *)&buffer[vectorIdx];
                uint8_t *bytePtr = (uint8_t *)&input[(blockOffset + vectorIdx) * vectorSize];
                for (int tmpIndex = 0; tmpIndex < vectorSize * sizeof(T); tmpIndex++)
                {
                    tmpCompressedDataGlobal[blockOffset * vectorSize * sizeof(T) + tmpOffset + tmpIndex] = *(bytePtr + tmpIndex);
                }
            }
            else
            {
                uint32_t tmpOffset = prefixBuffer[vectorIdx];
                tmpCompressedDataGlobal[blockOffset * vectorSize * sizeof(T) + tmpOffset] = lengthBuffer[vectorIdx];
                tmpCompressedDataGlobal[blockOffset * vectorSize * sizeof(T) + tmpOffset + 1] = offsetBuffer[vectorIdx];
            }
        }
    }

    // Copy the memeory back
    if (threadIdx.x == 0)
    {
        for (int flagArrIndex = 0; flagArrIndex < flagCount; flagArrIndex++)
        {
            tmpFlagArrGlobal[blockSize / 8 * blockIdx.x + flagArrIndex] = byteFlagArr[flagArrIndex];
        }
    }
}

// Define the compress Encode kernel functions
template <typename T>
__global__ void compressKernelIII(uint32_t numOfBlocks, uint32_t *flagArrOffsetGlobal, uint32_t *compressedDataOffsetGlobal, uint8_t *tmpFlagArrGlobal, uint8_t *tmpCompressedDataGlobal, uint8_t *flagArrGlobal, uint8_t *compressedDataGlobal)
{
    // Block size in uint of bytes
    const int blockSize = BLOCK_SIZE;

    // Window size in uint of bytes
    const int threadSize = THREAD_SIZE;

    const uint32_t vectorSize = VECTOR_SIZE;

    // find block index
    int blockIndex = blockIdx.x;

    int flagArrOffset = flagArrOffsetGlobal[blockIndex];
    int flagArrSize = flagArrOffsetGlobal[blockIndex + 1] - flagArrOffsetGlobal[blockIndex];

    int compressedDataOffset = compressedDataOffsetGlobal[blockIndex];
    int compressedDataSize = compressedDataOffsetGlobal[blockIndex + 1] - compressedDataOffsetGlobal[blockIndex];

    int tid = threadIdx.x;

    while (tid < flagArrSize)
    {
        flagArrGlobal[flagArrOffset + tid] = tmpFlagArrGlobal[blockSize / 8 * blockIndex + tid];
        tid += threadSize;
    }

    tid = threadIdx.x;

    while (tid < compressedDataSize)
    {
        compressedDataGlobal[compressedDataOffset + tid] = tmpCompressedDataGlobal[blockSize * blockIndex * sizeof(T) * vectorSize + tid];
        tid += threadSize;
    }
}

// Define the decompress kernel functions
template <typename T>
__global__ void decompressKernel(T *output, uint32_t numOfBlocks, uint32_t *flagArrOffsetGlobal, uint32_t *compressedDataOffsetGlobal, uint8_t *flagArrGlobal, uint8_t *compressedDataGlobal)
{
    // Block size in unit of datatype
    const uint32_t blockSize = BLOCK_SIZE;

    const uint32_t vectorSize = VECTOR_SIZE;

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < numOfBlocks)
    {
        int flagArrOffset = flagArrOffsetGlobal[tid];
        int flagArrSize = flagArrOffsetGlobal[tid + 1] - flagArrOffsetGlobal[tid];

        int compressedDataOffset = compressedDataOffsetGlobal[tid];

        uint32_t vectorIdx = 0;
        uint32_t compressedDataIndex = 0;

        uint8_t byteFlag;

        for (int flagArrayIndex = 0; flagArrayIndex < flagArrSize; flagArrayIndex++)
        {
            byteFlag = flagArrGlobal[flagArrOffset + flagArrayIndex];

            for (int bitCount = 0; bitCount < 8; bitCount++)
            {
                int matchFlag = (byteFlag >> bitCount) & 0x1;
                if (matchFlag == 1)
                {
                    int length = compressedDataGlobal[compressedDataOffset + compressedDataIndex];
                    int offset = compressedDataGlobal[compressedDataOffset + compressedDataIndex + 1];
                    compressedDataIndex += 2;
                    int vectorStart = vectorIdx;
                    for (int tmpDecompIndex = 0; tmpDecompIndex < length; tmpDecompIndex++)
                    {
                        for (int tmpSecondLevelIdx = 0; tmpSecondLevelIdx < vectorSize; tmpSecondLevelIdx++)
                        {
                            output[(tid * blockSize + vectorIdx) * vectorSize + tmpSecondLevelIdx] = output[(tid * blockSize + vectorStart - offset + tmpDecompIndex) * vectorSize + tmpSecondLevelIdx];
                        }
                        vectorIdx++;
                    }
                }
                else
                {
                    uint8_t *tmpPtr = (uint8_t *)&output[(tid * blockSize + vectorIdx) * vectorSize];
                    for (int tmpDecompIndex = 0; tmpDecompIndex < sizeof(T) * vectorSize; tmpDecompIndex++)
                    {
                        *(tmpPtr + tmpDecompIndex) = compressedDataGlobal[compressedDataOffset + compressedDataIndex + tmpDecompIndex];
                    }

                    compressedDataIndex += sizeof(T) * vectorSize;
                    vectorIdx++;
                }
                if (vectorIdx >= blockSize)
                {
                    return;
                }
            }
        }
    }
}

int main(int argc, char *argv[])
{
    std::string inputFileName;
    int decomp = 0;
    int opt;

    /* parse command line */
    while ((opt = getopt(argc, argv, "i:dh")) != -1)
    {
        switch (opt)
        {
        case 'i': /* input file name */
            inputFileName = optarg;
            break;

        case 'd': /* decompression */
            decomp = 1;
            break;

        case 'h': /* help */
            printf(" Usage for compression and decompression: ./gpulz -i {inputfile}\n");
            return 0;
        }
    }

    INPUT_TYPE *hostArray = io::read_binary_to_new_array<INPUT_TYPE>(inputFileName);

#ifdef DEBUG
    int debugOffset = 0;

    printf("print the first 1024 elements:\n");
    for (int tmpIndex = 0; tmpIndex < 1024; tmpIndex++)
    {
        std::cout << hostArray[tmpIndex + debugOffset] << "\t";
    }
    printf("\n");
#endif

    INPUT_TYPE *deviceArray;
    INPUT_TYPE *deviceOutput;
    uint32_t fileSize = io::FileSize(inputFileName);

    uint32_t *flagArrSizeGlobal;
    uint32_t *flagArrOffsetGlobal;
    uint32_t *compressedDataSizeGlobal;
    uint32_t *compressedDataOffsetGlobal;
    uint8_t *tmpFlagArrGlobal;
    uint8_t *tmpCompressedDataGlobal;
    uint8_t *flagArrGlobal;
    uint8_t *compressedDataGlobal;

    // calculate the padding size, unit in bytes
    uint32_t minimumChunkSize = BLOCK_SIZE * VECTOR_SIZE * sizeof(INPUT_TYPE);
    uint32_t paddingSize = fileSize % minimumChunkSize == 0 ? 0 : minimumChunkSize - fileSize % minimumChunkSize;

    // calculate the datatype size, unit in vector
    uint32_t inputVectorSize = static_cast<uint32_t>((fileSize + paddingSize) / sizeof(INPUT_TYPE) / VECTOR_SIZE);

    uint32_t numOfBlocks = inputVectorSize / BLOCK_SIZE;

    INPUT_TYPE *hostOutput = (INPUT_TYPE *)malloc(inputVectorSize * sizeof(INPUT_TYPE) * VECTOR_SIZE);

    // malloc the device buffer and set it as 0
    hipMalloc((void **)&deviceArray, fileSize + paddingSize);
    hipMalloc((void **)&deviceOutput, fileSize + paddingSize);

    hipMalloc((void **)&flagArrSizeGlobal, sizeof(uint32_t) * (numOfBlocks + 1));
    hipMalloc((void **)&flagArrOffsetGlobal, sizeof(uint32_t) * (numOfBlocks + 1));
    hipMalloc((void **)&compressedDataSizeGlobal, sizeof(uint32_t) * (numOfBlocks + 1));
    hipMalloc((void **)&compressedDataOffsetGlobal, sizeof(uint32_t) * (numOfBlocks + 1));
    hipMalloc((void **)&tmpFlagArrGlobal, sizeof(uint8_t) * inputVectorSize / 8);
    hipMalloc((void **)&tmpCompressedDataGlobal, sizeof(INPUT_TYPE) * inputVectorSize * VECTOR_SIZE);
    hipMalloc((void **)&flagArrGlobal, sizeof(uint8_t) * inputVectorSize / 8);
    hipMalloc((void **)&compressedDataGlobal, sizeof(INPUT_TYPE) * inputVectorSize * VECTOR_SIZE);

    // initialize the mem as 0
    hipMemset(deviceArray, 0, fileSize + paddingSize);
    hipMemset(deviceOutput, 0, fileSize + paddingSize);
    hipMemset(flagArrSizeGlobal, 0, sizeof(uint32_t) * (numOfBlocks + 1));
    hipMemset(flagArrOffsetGlobal, 0, sizeof(uint32_t) * (numOfBlocks + 1));
    hipMemset(compressedDataSizeGlobal, 0, sizeof(uint32_t) * (numOfBlocks + 1));
    hipMemset(compressedDataOffsetGlobal, 0, sizeof(uint32_t) * (numOfBlocks + 1));
    hipMemset(tmpFlagArrGlobal, 0, sizeof(uint8_t) * inputVectorSize / 8);
    hipMemset(tmpCompressedDataGlobal, 0, sizeof(INPUT_TYPE) * inputVectorSize * VECTOR_SIZE);

    // hipDeviceSetCacheConfig(hipFuncCachePreferShared);

    // copy host memory to device
    hipMemcpy(deviceArray, hostArray, fileSize, hipMemcpyHostToDevice);

    // printf("num of blocks: %d\nfile size: %d\npadding size: %d\n data type length: %d\n", numOfBlocks, fileSize, paddingSize, inputVectorSize);

    dim3 gridDim(numOfBlocks);
    dim3 blockDim(THREAD_SIZE);

    dim3 deGridDim(ceil(float(numOfBlocks) / 32));
    dim3 deBlockDim(32);

    uint32_t *flagArrOffsetGlobalHost;
    uint32_t *compressedDataOffsetGlobalHost;
    uint8_t *tmpFlagArrGlobalHost;
    uint8_t *tmpCompressedDataGlobalHost;
    uint8_t *flagArrGlobalHost;
    uint8_t *compressedDataGlobalHost;

    flagArrOffsetGlobalHost = (uint32_t *)malloc(sizeof(uint32_t) * (numOfBlocks + 1));
    compressedDataOffsetGlobalHost = (uint32_t *)malloc(sizeof(uint32_t) * (numOfBlocks + 1));
    tmpFlagArrGlobalHost = (uint8_t *)malloc(sizeof(uint8_t) * inputVectorSize / 8);
    tmpCompressedDataGlobalHost = (uint8_t *)malloc(sizeof(INPUT_TYPE) * inputVectorSize * VECTOR_SIZE);
    flagArrGlobalHost = (uint8_t *)malloc(sizeof(uint8_t) * inputVectorSize / 8);
    compressedDataGlobalHost = (uint8_t *)malloc(sizeof(INPUT_TYPE) * inputVectorSize * VECTOR_SIZE);

    hipEvent_t compStart, compStop, decompStart, decompStop;
    hipEventCreate(&compStart);
    hipEventCreate(&compStop);
    hipEventCreate(&decompStart);
    hipEventCreate(&decompStop);

    hipEventRecord(compStart);

    int minEncodeLength = 1;

    // launch kernels
    compressKernelI<INPUT_TYPE><<<gridDim, blockDim>>>(deviceArray, numOfBlocks, flagArrSizeGlobal, compressedDataSizeGlobal, tmpFlagArrGlobal, tmpCompressedDataGlobal, minEncodeLength);

    // Determine temporary device storage requirements
    void *flag_d_temp_storage = NULL;
    size_t flag_temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(flag_d_temp_storage, flag_temp_storage_bytes, flagArrSizeGlobal, flagArrOffsetGlobal, numOfBlocks + 1);

    // Allocate temporary storage
    hipMalloc(&flag_d_temp_storage, flag_temp_storage_bytes);

    // Run exclusive prefix sum
    hipcub::DeviceScan::ExclusiveSum(flag_d_temp_storage, flag_temp_storage_bytes, flagArrSizeGlobal, flagArrOffsetGlobal, numOfBlocks + 1);

    // Determine temporary device storage requirements
    void *data_d_temp_storage = NULL;
    size_t data_temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(data_d_temp_storage, data_temp_storage_bytes, compressedDataSizeGlobal, compressedDataOffsetGlobal, numOfBlocks + 1);

    // Allocate temporary storage
    hipMalloc(&data_d_temp_storage, data_temp_storage_bytes);

    // Run exclusive prefix sum
    hipcub::DeviceScan::ExclusiveSum(data_d_temp_storage, data_temp_storage_bytes, compressedDataSizeGlobal, compressedDataOffsetGlobal, numOfBlocks + 1);

    compressKernelIII<INPUT_TYPE><<<gridDim, blockDim>>>(numOfBlocks, flagArrOffsetGlobal, compressedDataOffsetGlobal, tmpFlagArrGlobal, tmpCompressedDataGlobal, flagArrGlobal, compressedDataGlobal);

    hipEventRecord(compStop);

    hipEventRecord(decompStart);

    decompressKernel<INPUT_TYPE><<<deGridDim, deBlockDim>>>(deviceOutput, numOfBlocks, flagArrOffsetGlobal, compressedDataOffsetGlobal, flagArrGlobal, compressedDataGlobal);
    hipEventRecord(decompStop);

    // copy the memory back to global
    hipMemcpy(flagArrOffsetGlobalHost, flagArrOffsetGlobal, sizeof(uint32_t) * (numOfBlocks + 1), hipMemcpyDeviceToHost);
    hipMemcpy(compressedDataOffsetGlobalHost, compressedDataOffsetGlobal, sizeof(uint32_t) * (numOfBlocks + 1), hipMemcpyDeviceToHost);
    hipMemcpy(tmpFlagArrGlobalHost, tmpFlagArrGlobal, sizeof(uint8_t) * inputVectorSize / 8, hipMemcpyDeviceToHost);
    hipMemcpy(tmpCompressedDataGlobalHost, tmpCompressedDataGlobal, sizeof(INPUT_TYPE) * inputVectorSize, hipMemcpyDeviceToHost);
    hipMemcpy(flagArrGlobalHost, flagArrGlobal, sizeof(uint8_t) * inputVectorSize / 8, hipMemcpyDeviceToHost);
    hipMemcpy(compressedDataGlobalHost, compressedDataGlobal, sizeof(INPUT_TYPE) * inputVectorSize, hipMemcpyDeviceToHost);

    hipMemcpy(hostOutput, deviceOutput, fileSize, hipMemcpyDeviceToHost);

#ifdef DEBUG
    printf("print the first 1024 flag array offset elements:\n");
    for (int tmpIndex = 0; tmpIndex < 1024; tmpIndex++)
    {
        printf("%d\t", flagArrOffsetGlobalHost[tmpIndex]);
    }
    printf("\n");

    printf("print the first 1024 compressed data offset elements:\n");
    for (int tmpIndex = 0; tmpIndex < 1024; tmpIndex++)
    {
        printf("%d\t", compressedDataOffsetGlobalHost[tmpIndex]);
    }
    printf("\n");

    printf("print the first 1024 flag array elements:\n");
    for (int tmpIndex = 0; tmpIndex < 1024; tmpIndex++)
    {
        printf("%d\t", flagArrGlobalHost[tmpIndex]);
    }
    printf("\n");

    printf("print the first 1024 compressed data elements:\n");
    for (int tmpIndex = 0; tmpIndex < 1024; tmpIndex++)
    {
        printf("%d\t", compressedDataGlobalHost[tmpIndex]);
    }
    printf("\n");

    // printf("print the first 1024 tmp flag array elements:\n");
    // for (int tmpIndex = 0; tmpIndex < 1024; tmpIndex++)
    // {
    //   printf("%d\t", tmpFlagArrGlobalHost[tmpIndex]);
    // }
    // printf("\n");
#endif

    // verify the final output
    for (int verifyIndex = 0; verifyIndex < fileSize / sizeof(INPUT_TYPE); verifyIndex++)
    {
        if (hostArray[verifyIndex] != hostOutput[verifyIndex])
        {
            printf("verification failed!!! Index %d is wrong\n", verifyIndex);
            std::cout << "hostArray: " << hostArray[verifyIndex] << ", hostOutput: " << hostOutput[verifyIndex] << std::endl;
            break;
        }
    }

    float originalSize = fileSize;
    float compressedSize = sizeof(uint32_t) * (numOfBlocks + 1) * 2 + flagArrOffsetGlobalHost[numOfBlocks] + compressedDataOffsetGlobalHost[numOfBlocks];
    float compressionRatio = originalSize / compressedSize;
    std::cout << "compression ratio: " << compressionRatio << std::endl;

    hipEventSynchronize(decompStop);
    float compTime = 0;
    float decompTime = 0;
    hipEventElapsedTime(&compTime, compStart, compStop);
    hipEventElapsedTime(&decompTime, decompStart, decompStop);
    float compTp = float(fileSize) / 1024 / 1024 / compTime;
    float decompTp = float(fileSize) / 1024 / 1024 / decompTime;
    std::cout << "compression e2e throughput: " << compTp << " GB/s" << std::endl;
    std::cout << "decompression e2e throughput: " << decompTp << " GB/s" << std::endl;

    // free dynamic arrays
    free(flagArrOffsetGlobalHost);
    free(compressedDataOffsetGlobalHost);
    free(tmpFlagArrGlobalHost);
    free(tmpCompressedDataGlobalHost);
    free(flagArrGlobalHost);
    free(compressedDataGlobalHost);

    hipFree(deviceArray);
    hipFree(deviceOutput);

    hipFree(flagArrSizeGlobal);
    hipFree(flagArrOffsetGlobal);
    hipFree(compressedDataSizeGlobal);
    hipFree(compressedDataOffsetGlobal);
    hipFree(tmpFlagArrGlobal);
    hipFree(tmpCompressedDataGlobal);
    hipFree(flagArrGlobal);
    hipFree(compressedDataGlobal);

    hipFree(flag_d_temp_storage);
    hipFree(data_d_temp_storage);

    free(hostOutput);

    delete hostArray;

    return 0;
}